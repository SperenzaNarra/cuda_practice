#include "hip/hip_runtime.h"
#include "common.cuh"
#include <math.h>

void write_pixels(std::ostream &out, vec3double *pixels)
{
    for (int i = 0; i < IMAGE_WIDTH * IMAGE_HEIGHT; i++)
    {
        vec3double *pixel = &pixels[i];
        int r = static_cast<int>(255.999 * std::sqrt(pixel->r));
        int g = static_cast<int>(255.999 * std::sqrt(pixel->g));
        int b = static_cast<int>(255.999 * std::sqrt(pixel->b));
        out << r << ' ' << g << ' ' << b << std::endl;
    }
}

__device__ double random_double(double min, double max, hiprandState &rand_state)
{
    return min + (max-min) * hiprand_uniform_double(&rand_state);
}

__device__ vec3double get_default_color(vec3double &direction)
{
    float t = 0.5 * (direction.y + 1.0);
    return (1.0 - t) * vec3double(1.0) + t * vec3double(0.5, 0.7, 1.0);
}

__device__ vec3double get_rand_hemisphere(vec3double &normal, hiprandState &rand_state)
{
    double angle = 2 * M_PI * hiprand_uniform_double(&rand_state);
    double rad_2 = hiprand_uniform_double(&rand_state);
    double rad = sqrt(rad_2);
    vec3double u = (cross(abs(normal.x) > .1 ? vec3double(0, 1, 0) : vec3double(1, 0, 0), normal)).normalized(),
               v = cross(normal, u);
    // sample unit hemisphere
    vec3double d = (u * cos(angle) * rad + v * sin(angle) * rad + normal * sqrt(1 - rad_2)).normalized();
    return d;
}

__device__ vec3double refract(vec3double &in_direction, vec3double &normal, double etai_over_etat)
{
    double cos_theta = min(dot(-in_direction, normal), 1.0);
    vec3double out_para = etai_over_etat * (in_direction + cos_theta * normal);
    vec3double out_perp = -sqrt(abs(1.0 - out_para.length2())) * normal;
    return out_para + out_perp;
}

__device__ vec3double get_color(ray &in_ray, sphere* spheres, int sphere_size, hiprandState &rand_state)
{
    vec3double attenuation = vec3double(1);
    int depth = 0;
    // while (true)
    while (depth < 10)
    {
        sphere* target_sphere = NULL;
        double t = RAY_T_MAX;
        ray normal;
        bool into;

        for (int i = 0; i < sphere_size; i++)
        {
            if (spheres[i].hit(in_ray, normal, into, t))
            {
                target_sphere = &spheres[i];
            }
        }

        if (!target_sphere) return attenuation * get_default_color(in_ray.direction);

        double p = max(attenuation.r, max(attenuation.g, attenuation.b));
        if (++depth > 5)
            if (hiprand_uniform_double(&rand_state) < p) //R.R.
                attenuation = attenuation * (1 / p);
            else {
                break;
            }

        // update
        attenuation *= target_sphere->color;
        in_ray.origin = normal.origin;

        switch (target_sphere->type)
        {
        case MATERIAL_LAMBERTIAN:
            in_ray.direction = get_rand_hemisphere(normal.direction, rand_state);
            break;
        case MATERIAL_METAL:
            in_ray.direction = in_ray.direction - 2 * dot(in_ray.direction, normal.direction) * normal.direction;
            in_ray.direction += get_rand_hemisphere(in_ray.direction, rand_state) * target_sphere->fuzz;
            break;
        case MATERIAL_DIELECTRIC:{
            double n_air = 1.0, n_glass = target_sphere->ir;
            double n_ratio = into ? n_air / n_glass : n_glass / n_air;
            double d_dot_n = dot(in_ray.direction, normal.direction),
                   cos2t = 1 - square(n_ratio) * (1 - square(d_dot_n));
            if (cos2t < 0) {   // Total internal reflection
                in_ray.direction = in_ray.direction - 2 * dot(in_ray.direction, normal.direction) * normal.direction;
                break;
            }

            vec3double tdir = (in_ray.direction * n_ratio - normal.direction * (d_dot_n * n_ratio + sqrt(cos2t))).normalized();

            double refl_norm = square(n_glass - n_air) / square(n_glass + n_air),
                   c = 1 - (into ? -d_dot_n : -dot(tdir, normal.direction));
            double refl_fresnel = refl_norm + (1 - refl_norm) * c * c * c * c * c,
                   trans_fresnel = 1 - refl_fresnel,
                   prob_refl = .25 + .5 * refl_fresnel;

            if (hiprand_uniform_double(&rand_state) < prob_refl) { // Russian roulette
                attenuation = attenuation * (refl_fresnel / prob_refl);
                in_ray.direction = in_ray.direction - 2 * dot(in_ray.direction, normal.direction) * normal.direction;
            } else {
                attenuation = attenuation * (trans_fresnel / (1 - prob_refl));
                in_ray.direction = tdir;
            }
            break;
        }

        default:
            return vec3double(0);
        }
        in_ray.direction = in_ray.direction.normalized();
    }
    
    // out of depth
    return vec3double(0);
}

__global__ void render(vec3double *pixels, camera** camera, sphere* spheres, int sphere_size)
{
    int col = threadIdx.x + blockIdx.x * blockDim.x;
    int row = threadIdx.y + blockIdx.y * blockDim.y;
    if (col >= IMAGE_WIDTH || row >= IMAGE_HEIGHT) return;

    int index = col + row * IMAGE_WIDTH;
    row = IMAGE_HEIGHT - row - 1;

    hiprandState rand_state;
    hiprand_init((unsigned long long)clock64() + index, 0, 0, &rand_state);

    for (int s = 0; s < SAMPLES_PER_PIXEL; s++)
    {
        double u = ((double) col + hiprand_uniform_double(&rand_state)) / (IMAGE_WIDTH - 1);
        double v = ((double) row + hiprand_uniform_double(&rand_state)) / (IMAGE_HEIGHT - 1);
        ray in_ray = (*camera)->get_ray(u, v);
        pixels[index] += get_color(in_ray, spheres, sphere_size, rand_state);
    }
    pixels[index] /= SAMPLES_PER_PIXEL;
}