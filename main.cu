#include "hip/hip_runtime.h"
#include "common.cuh"
#include <vector>

#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )
void check_cuda(hipError_t result, char const *const func, const char *const file, int const line) 
{
    if (result) {
        std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " <<
        file << ":" << line << " '" << func << "' \n";
        // Make sure we call CUDA Device Reset before exiting
        hipDeviceReset();
        exit(99);
    }
}

__global__ void render_init(camera** d_camera)
{
    *d_camera = new camera(2.0, 1.0);
}

__global__ void render_free(camera** d_camera)
{
    delete *d_camera;
}

int main()
{
    dim3 dimBlock(BLOCK_SIZE);
    dim3 dimGrid(IMAGE_WIDTH/BLOCK_SIZE+1);

    // buffer
    int pixel_size = sizeof(vec3double) * IMAGE_WIDTH;
    vec3double  h_pixels[IMAGE_WIDTH];
    vec3double *d_pixels;
    checkCudaErrors(hipMallocManaged((void**)&d_pixels, pixel_size));

    // camera
    camera** d_camera;
    checkCudaErrors(hipMallocManaged((void**)&d_camera, sizeof(camera*)));

    // spheres
    std::vector<sphere> h_spheres;
    // h_spheres.push_back(sphere(vec3double(0, -100.5, -1), 100, vec3double(0.5)));
    // h_spheres.push_back(sphere(vec3double(0, 0, -1), 0.5, vec3double(0.5)));

    h_spheres.push_back(sphere(vec3double(0, -100.5, -1), 100, vec3double(0.8, 0.8, 0)));
    h_spheres.push_back(sphere(vec3double(0, 0, -1), 0.5, vec3double(0.7, 0.3, 0.3)));
    h_spheres.push_back(sphere(vec3double(-1, 0, -1), 0.5, vec3double(0.8), MATERIAL_METAL));
    h_spheres.push_back(sphere(vec3double(1, 0, -1), 0.5, vec3double(0.8, 0.6, 0.2), MATERIAL_METAL));

    sphere* d_spheres;
    if (h_spheres.size())
    {
        checkCudaErrors(hipMallocManaged((void**)&d_spheres, sizeof(sphere) * h_spheres.size()));
        checkCudaErrors(hipMemcpy(d_spheres, &h_spheres[0], sizeof(sphere) * h_spheres.size(), hipMemcpyHostToDevice));
    }

    // init
    render_init<<<1, 1>>>(d_camera);

    // core
    std::cout << "P3\n" << IMAGE_WIDTH << ' ' << IMAGE_HEIGHT << "\n255\n";
    for (int row = IMAGE_HEIGHT - 1; row >= 0; row--)
    {
        std::cerr << "\rScanlines remaining: " << row << ' ' << std::flush;
        render<<<dimGrid, dimBlock>>>(d_pixels, row, d_camera, d_spheres, h_spheres.size());

        checkCudaErrors(hipGetLastError());
        checkCudaErrors(hipDeviceSynchronize());

        checkCudaErrors(hipMemcpy(h_pixels, d_pixels, pixel_size, hipMemcpyDeviceToHost));
        write_pixels(std::cout, h_pixels);
    }
    std::cerr << "\nDone.\n";

    render_free<<<1, 1>>>(d_camera);
    checkCudaErrors(hipFree(d_pixels));
    checkCudaErrors(hipFree(d_camera));
    if (h_spheres.size())
        checkCudaErrors(hipFree(d_spheres));
}